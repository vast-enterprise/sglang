#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>

#include <cmath>
#include <cub/block/block_reduce.cuh>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

template <typename T>
__global__ void per_token_quant_fp8_kernel(
    const T* __restrict__ input,
    FP8_TYPE* __restrict__ output_q,
    float* __restrict__ output_s,
    const int64_t hidden_dim,
    const int64_t num_tokens) {
  const int token_idx = blockIdx.x;

  if (token_idx >= num_tokens) return;

  const int tid = threadIdx.x;
  const int block_dim = blockDim.x;

  const T* token_input = input + token_idx * hidden_dim;
  FP8_TYPE* token_output = output_q + token_idx * hidden_dim;

  float max_value = 0.0f;

  for (int i = tid; i < hidden_dim; i += block_dim) {
    float val = static_cast<float>(token_input[i]);
    max_value = fmaxf(max_value, fabsf(val));
  }

  max_value = blockReduceMax(max_value);

  __shared__ float block_max;
  if (tid == 0) {
    block_max = max_value / FP8_E4M3_MAX;
    output_s[token_idx] = block_max;
  }
  __syncthreads();

  const float scale_val = 1.0f / block_max;

  constexpr uint32_t vec_size = 16 / sizeof(T);
  using vec_t = flashinfer::vec_t<T, vec_size>;

  const int32_t num_vec_elems = hidden_dim / vec_size;

  for (int32_t i = tid; i < num_vec_elems; i += block_dim) {
    vec_t input_vec;
    input_vec.cast_load(token_input + i * vec_size);

    FP8_TYPE output_arr[vec_size];
#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      float val = fmax(fmin(static_cast<float>(input_vec[j]) * scale_val, FP8_E4M3_MAX), -FP8_E4M3_MAX);
#ifndef USE_ROCM
      output_arr[j] = static_cast<FP8_TYPE>(val);
#else
      output_arr[j] = c10::Float8_e4m3fnuz(
          __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
          c10::Float8_e4m3fnuz::from_bits());
#endif
    }

#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      token_output[i * vec_size + j] = output_arr[j];
    }
  }

  const int32_t remaining_start = num_vec_elems * vec_size;
  for (int32_t idx = remaining_start + tid; idx < hidden_dim; idx += block_dim) {
    float val = fmax(-FP8_E4M3_MAX, fmin(static_cast<float>(token_input[idx]) * scale_val, FP8_E4M3_MAX));
#ifndef USE_ROCM
    token_output[idx] = static_cast<FP8_TYPE>(val);
#else
    token_output[idx] = c10::Float8_e4m3fnuz(
        __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
        c10::Float8_e4m3fnuz::from_bits());
#endif
  }
}

void sgl_per_token_quant_fp8(torch::Tensor input, torch::Tensor output_q, torch::Tensor output_s) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);

  const auto input_sizes = input.sizes();
  const int64_t num_tokens = input_sizes[0];
  const int64_t hidden_dim = input_sizes[1];

  const int block_size = 128;
  const int num_blocks = num_tokens;

  dim3 grid(num_blocks);
  dim3 block(block_size);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    per_token_quant_fp8_kernel<scalar_t><<<grid, block, 0, stream>>>(
        static_cast<scalar_t*>(input.data_ptr()),
        static_cast<FP8_TYPE*>(output_q.data_ptr()),
        static_cast<float*>(output_s.data_ptr()),
        hidden_dim,
        num_tokens);
    return true;
  });
}
